#include "hip/hip_runtime.h"
﻿// -----------------------------------------------------------------------------------------
// NVEnc by rigaya
// -----------------------------------------------------------------------------------------
//
// The MIT License
//
// Copyright (c) 2014-2016 rigaya
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
// THE SOFTWARE.
//
// ------------------------------------------------------------------------------------------

#include <map>
#include <array>
#define _USE_MATH_DEFINES
#include <cmath>
#include <limits>
#include "convert_csp.h"
#include "NVEncFilterEdgelevel.h"
#include "rgy_prm.h"
#pragma warning (push)
#pragma warning (disable: 4819)
#include "hip/hip_runtime.h"
#include ""
#pragma warning (pop)
#include "rgy_cuda_util_kernel.h"

static const int EDGELEVEL_BLOCK_X = 32;
static const int EDGELEVEL_BLOCK_Y = 16;

__device__ __inline__
void check_min_max(float& min, float& max, float value) {
    max = fmaxf(max, value);
    min = fminf(min, value);
}

template<typename Type, int bit_depth>
__global__ void kernel_edgelevel(uint8_t *__restrict__ pDst, const int dstPitch, const int dstWidth, const int dstHeight,
    hipTextureObject_t texSrc, const float strength, const float threshold, const float black, const float white) {
    const int ix = blockIdx.x * blockDim.x + threadIdx.x;
    const int iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix < dstWidth && iy < dstHeight) {
        float x = ix + 0.5f;
        float y = iy + 0.5f;

        float center = tex2D<float>(texSrc, x, y);
        float min = center;
        float vmin = center;
        float max = center;
        float vmax = center;

        check_min_max(min,  max,  tex2D<float>(texSrc, x - 2.0f, y));
        check_min_max(vmin, vmax, tex2D<float>(texSrc, x, y - 2.0f));
        check_min_max(min,  max,  tex2D<float>(texSrc, x - 1.0f, y));
        check_min_max(vmin, vmax, tex2D<float>(texSrc, x, y - 1.0f));
        check_min_max(min,  max,  tex2D<float>(texSrc, x + 1.0f, y));
        check_min_max(vmin, vmax, tex2D<float>(texSrc, x, y + 1.0f));
        check_min_max(min,  max,  tex2D<float>(texSrc, x + 2.0f, y));
        check_min_max(vmin, vmax, tex2D<float>(texSrc, x, y + 2.0f));

        if (max - min < vmax - vmin) {
            max = vmax, min = vmin;
        }

        if (max - min > threshold) {
            float avg = (min + max) * 0.5f;
            if (center == min)
                min -= black;
            min -= black;
            if (center == max)
                max += white;
            max += white;

            center = fminf(fmaxf((center + ((center - avg) * strength)), min), max);
        }

        Type *ptr = (Type *)(pDst + iy * dstPitch + ix * sizeof(Type));
        ptr[0] = (Type)(clamp(center, 0.0f, 1.0f - RGY_FLT_EPS) * ((1 << bit_depth) - 1));
    }
}

template<typename Type>
hipError_t textureCreateEdgelevel(hipTextureObject_t &tex, hipTextureFilterMode filterMode, hipTextureReadMode readMode, uint8_t *ptr, int pitch, int width, int height) {
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypePitch2D;
    resDesc.res.pitch2D.devPtr = ptr;
    resDesc.res.pitch2D.pitchInBytes = pitch;
    resDesc.res.pitch2D.width = width;
    resDesc.res.pitch2D.height = height;
    resDesc.res.pitch2D.desc = hipCreateChannelDesc<Type>();

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = filterMode;
    texDesc.readMode = readMode;
    texDesc.normalizedCoords = 0;

    return hipCreateTextureObject(&tex, &resDesc, &texDesc, nullptr);
}

template<typename Type, int bit_depth>
static RGY_ERR edgelevel_plane(RGYFrameInfo *pOutputFrame, const RGYFrameInfo *pInputFrame,
    float strength, float threshold, float black, float white, hipStream_t stream) {
    dim3 blockSize(EDGELEVEL_BLOCK_X, EDGELEVEL_BLOCK_Y);
    dim3 gridSize(divCeil(pOutputFrame->width, blockSize.x), divCeil(pOutputFrame->height, blockSize.y));
    strength  /= (1<<4);
    threshold /= (1<<((sizeof(Type) * 8) - 1));
    black     /= (1<<(sizeof(Type) * 8));
    white     /= (1<<(sizeof(Type) * 8));

    hipTextureObject_t texSrc = 0;
    auto cudaerr = textureCreateEdgelevel<Type>(texSrc, hipFilterModePoint, hipReadModeNormalizedFloat, pInputFrame->ptr[0], pInputFrame->pitch[0], pInputFrame->width, pInputFrame->height);
    if (cudaerr != hipSuccess) {
        return err_to_rgy(cudaerr);
    }
    kernel_edgelevel<Type, bit_depth><<<gridSize, blockSize, 0, stream>>>((uint8_t *)pOutputFrame->ptr[0],
        pOutputFrame->pitch[0], pOutputFrame->width, pOutputFrame->height,
        texSrc, strength, threshold, black, white);
    cudaerr = hipGetLastError();
    if (cudaerr != hipSuccess) {
        return err_to_rgy(cudaerr);
    }
    cudaerr = hipDestroyTextureObject(texSrc);
    if (cudaerr != hipSuccess) {
    }
    return RGY_ERR_NONE;
}

template<typename Type, int bit_depth>
static RGY_ERR edgelevel_frame(RGYFrameInfo *pOutputFrame, const RGYFrameInfo *pInputFrame,
    float strength, float threshold, float black, float white, hipStream_t stream) {
    const auto planeInputY = getPlane(pInputFrame, RGY_PLANE_Y);
    const auto planeInputU = getPlane(pInputFrame, RGY_PLANE_U);
    const auto planeInputV = getPlane(pInputFrame, RGY_PLANE_V);
    auto planeOutputY = getPlane(pOutputFrame, RGY_PLANE_Y);
    auto planeOutputU = getPlane(pOutputFrame, RGY_PLANE_U);
    auto planeOutputV = getPlane(pOutputFrame, RGY_PLANE_V);
    auto err = edgelevel_plane<Type, bit_depth>(&planeOutputY, &planeInputY,
        strength,
        threshold,
        black,
        white,
        stream);
    if (err != RGY_ERR_NONE) {
        return err;
    }
    err = copyPlane(&planeOutputU, &planeInputU, stream);
    if (err != RGY_ERR_NONE) {
        return err;
    }
    err = copyPlane(&planeOutputV, &planeInputV, stream);
    if (err != RGY_ERR_NONE) {
        return err;
    }
    auto cudaerr = hipGetLastError();
    if (cudaerr != hipSuccess) {
        return err_to_rgy(cudaerr);
    }
    return RGY_ERR_NONE;
}

NVEncFilterEdgelevel::NVEncFilterEdgelevel() {
    m_name = _T("edgelevel");
}

NVEncFilterEdgelevel::~NVEncFilterEdgelevel() {
    close();
}

RGY_ERR NVEncFilterEdgelevel::init(shared_ptr<NVEncFilterParam> pParam, shared_ptr<RGYLog> pPrintMes) {
    RGY_ERR sts = RGY_ERR_NONE;
    m_pLog = pPrintMes;
    auto pEdgelevelParam = std::dynamic_pointer_cast<NVEncFilterParamEdgelevel>(pParam);
    if (!pEdgelevelParam) {
        AddMessage(RGY_LOG_ERROR, _T("Invalid parameter type.\n"));
        return RGY_ERR_INVALID_PARAM;
    }
    //パラメータチェック
    if (pEdgelevelParam->frameOut.height <= 0 || pEdgelevelParam->frameOut.width <= 0) {
        AddMessage(RGY_LOG_ERROR, _T("Invalid parameter.\n"));
        return RGY_ERR_INVALID_PARAM;
    }
    if (pEdgelevelParam->edgelevel.strength < -31.0f || 31.0f < pEdgelevelParam->edgelevel.strength) {
        pEdgelevelParam->edgelevel.strength = clamp(pEdgelevelParam->edgelevel.strength, -31.0f, 31.0f);
        AddMessage(RGY_LOG_WARN, _T("strength should be in range of %.1f - %.1f.\n"), -31.0f, 31.0f);
    }
    if (pEdgelevelParam->edgelevel.threshold < 0.0f || 255.0f < pEdgelevelParam->edgelevel.threshold) {
        pEdgelevelParam->edgelevel.threshold = clamp(pEdgelevelParam->edgelevel.threshold, 0.0f, 255.0f);
        AddMessage(RGY_LOG_WARN, _T("threshold should be in range of %.1f - %.1f.\n"), 0.0f, 255.0f);
    }
    if (pEdgelevelParam->edgelevel.black < 0.0f || 31.0f < pEdgelevelParam->edgelevel.black) {
        pEdgelevelParam->edgelevel.black = clamp(pEdgelevelParam->edgelevel.black, 0.0f, 31.0f);
        AddMessage(RGY_LOG_WARN, _T("black should be in range of %.1f - %.1f.\n"), 0.0f, 31.0f);
    }
    if (pEdgelevelParam->edgelevel.white < 0.0f || 31.0f < pEdgelevelParam->edgelevel.white) {
        pEdgelevelParam->edgelevel.white = clamp(pEdgelevelParam->edgelevel.white, 0.0f, 31.0f);
        AddMessage(RGY_LOG_WARN, _T("white should be in range of %.1f - %.1f.\n"), 0.0f, 31.0f);
    }

    sts = AllocFrameBuf(pEdgelevelParam->frameOut, 1);
    if (sts != RGY_ERR_NONE) {
        AddMessage(RGY_LOG_ERROR, _T("failed to allocate memory: %s.\n"), get_err_mes(sts));
        return RGY_ERR_MEMORY_ALLOC;
    }
    for (int i = 0; i < RGY_CSP_PLANES[pParam->frameOut.csp]; i++) {
        pEdgelevelParam->frameOut.pitch[i] = m_frameBuf[0]->frame.pitch[i];
    }

    setFilterInfo(pParam->print());
    m_param = pEdgelevelParam;
    return sts;
}

tstring NVEncFilterParamEdgelevel::print() const {
    return edgelevel.print();
}

RGY_ERR NVEncFilterEdgelevel::run_filter(const RGYFrameInfo *pInputFrame, RGYFrameInfo **ppOutputFrames, int *pOutputFrameNum, hipStream_t stream) {
    RGY_ERR sts = RGY_ERR_NONE;
    if (pInputFrame->ptr[0] == nullptr) {
        return sts;
    }

    *pOutputFrameNum = 1;
    if (ppOutputFrames[0] == nullptr) {
        auto pOutFrame = m_frameBuf[m_nFrameIdx].get();
        ppOutputFrames[0] = &pOutFrame->frame;
        m_nFrameIdx = (m_nFrameIdx + 1) % m_frameBuf.size();
    }
    ppOutputFrames[0]->picstruct = pInputFrame->picstruct;
    if (interlaced(*pInputFrame)) {
        return filter_as_interlaced_pair(pInputFrame, ppOutputFrames[0], hipStreamDefault);
    }
    const auto memcpyKind = getCudaMemcpyKind(pInputFrame->mem_type, ppOutputFrames[0]->mem_type);
    if (memcpyKind != hipMemcpyDeviceToDevice) {
        AddMessage(RGY_LOG_ERROR, _T("only supported on device memory.\n"));
        return RGY_ERR_INVALID_PARAM;
    }
    if (m_param->frameOut.csp != m_param->frameIn.csp) {
        AddMessage(RGY_LOG_ERROR, _T("csp does not match.\n"));
        return RGY_ERR_INVALID_PARAM;
    }
    auto pEdgelevelParam = std::dynamic_pointer_cast<NVEncFilterParamEdgelevel>(m_param);
    if (!pEdgelevelParam) {
        AddMessage(RGY_LOG_ERROR, _T("Invalid parameter type.\n"));
        return RGY_ERR_INVALID_PARAM;
    }

    static const std::map<RGY_CSP, decltype(edgelevel_frame<uint8_t, 8>)*> denoise_list = {
        { RGY_CSP_YV12,      edgelevel_frame<uint8_t,   8> },
        { RGY_CSP_YV12_16,   edgelevel_frame<uint16_t, 16> },
        { RGY_CSP_YUV444,    edgelevel_frame<uint8_t,   8> },
        { RGY_CSP_YUV444_16, edgelevel_frame<uint16_t, 16> }
    };
    if (denoise_list.count(pInputFrame->csp) == 0) {
        AddMessage(RGY_LOG_ERROR, _T("unsupported csp %s.\n"), RGY_CSP_NAMES[pInputFrame->csp]);
        return RGY_ERR_UNSUPPORTED;
    }
    sts = denoise_list.at(pInputFrame->csp)(ppOutputFrames[0], pInputFrame,
        pEdgelevelParam->edgelevel.strength,
        pEdgelevelParam->edgelevel.threshold,
        pEdgelevelParam->edgelevel.black,
        pEdgelevelParam->edgelevel.white,
        stream);
    if (sts != RGY_ERR_NONE) {
        AddMessage(RGY_LOG_ERROR, _T("error at edgelevel(%s): %s.\n"),
            RGY_CSP_NAMES[pInputFrame->csp],
            get_err_mes(sts));
        return sts;
    }
    return sts;
}

void NVEncFilterEdgelevel::close() {
    m_frameBuf.clear();
}
