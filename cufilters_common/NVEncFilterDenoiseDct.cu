#include "hip/hip_runtime.h"
﻿// -----------------------------------------------------------------------------------------
// NVEnc by rigaya
// -----------------------------------------------------------------------------------------
//
// The MIT License
//
// Copyright (c) 2014-2016 rigaya
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
// THE SOFTWARE.
//
// ------------------------------------------------------------------------------------------

#include <array>
#include <map>
#include "convert_csp.h"
#include "NVEncFilterDenoiseDct.h"
#include "rgy_prm.h"
#pragma warning (push)
#pragma warning (disable: 4819)
#include "hip/hip_runtime.h"
#include ""
#include "rgy_cuda_util_kernel.h"
#pragma warning (pop)

#define DENOISE_BLOCK_SIZE_X (8) //ひとつのスレッドブロックの担当するx方向の8x8ブロックの数

#define DENOISE_SHARED_BLOCK_NUM_X (DENOISE_BLOCK_SIZE_X+2) //sharedメモリ上のx方向の8x8ブロックの数
#define DENOISE_SHARED_BLOCK_NUM_Y (2)                      //sharedメモリ上のy方向の8x8ブロックの数

#define DENOISE_LOOP_COUNT_BLOCK (8)

#define DCT3X3_0_0 ( 0.5773502691896258f) /*  1/sqrt(3) */
#define DCT3X3_0_1 ( 0.5773502691896258f) /*  1/sqrt(3) */
#define DCT3X3_0_2 ( 0.5773502691896258f) /*  1/sqrt(3) */
#define DCT3X3_1_0 ( 0.7071067811865475f) /*  1/sqrt(2) */
#define DCT3X3_1_2 (-0.7071067811865475f) /* -1/sqrt(2) */
#define DCT3X3_2_0 ( 0.4082482904638631f) /*  1/sqrt(6) */
#define DCT3X3_2_1 (-0.8164965809277261f) /* -2/sqrt(6) */
#define DCT3X3_2_2 ( 0.4082482904638631f) /*  1/sqrt(6) */

//CUDA Sampleより拝借
#define C_a 1.387039845322148f //!< a = (2^0.5) * cos(    pi / 16);  Used in forward and inverse DCT.
#define C_b 1.306562964876377f //!< b = (2^0.5) * cos(    pi /  8);  Used in forward and inverse DCT.
#define C_c 1.175875602419359f //!< c = (2^0.5) * cos(3 * pi / 16);  Used in forward and inverse DCT.
#define C_d 0.785694958387102f //!< d = (2^0.5) * cos(5 * pi / 16);  Used in forward and inverse DCT.
#define C_e 0.541196100146197f //!< e = (2^0.5) * cos(3 * pi /  8);  Used in forward and inverse DCT.
#define C_f 0.275899379282943f //!< f = (2^0.5) * cos(7 * pi / 16);  Used in forward and inverse DCT.

//Normalization constant that is used in forward and inverse DCT
#define C_norm 0.3535533905932737f // 1 / (8^0.5)

template<typename T> __device__ __inline__ T setval(float val);
template<> __device__ __inline__ float setval(float val) { return val; };
#if ENABLE_CUDA_FP16_HOST
template<> __device__ __inline__ __half2 setval(float val) { return __float2half2_rn(val); }
__device__ __inline__
__half2 fabs(__half2 val) {
    __half2 h;
    RGY_HALF2_TO_UI(h) = RGY_HALF2_TO_UI(val) & 0x7fff7fffu;
    return h;
}
#endif //#if ENABLE_CUDA_FP16_HOST

template<typename T, int Step>
__device__ void CUDAsubroutineInplaceDCT8vector(T *Vect0) {
    T *Vect1 = Vect0 + Step;
    T *Vect2 = Vect1 + Step;
    T *Vect3 = Vect2 + Step;
    T *Vect4 = Vect3 + Step;
    T *Vect5 = Vect4 + Step;
    T *Vect6 = Vect5 + Step;
    T *Vect7 = Vect6 + Step;

    T X07P = (*Vect0) + (*Vect7);
    T X16P = (*Vect1) + (*Vect6);
    T X25P = (*Vect2) + (*Vect5);
    T X34P = (*Vect3) + (*Vect4);

    T X07M = (*Vect0) - (*Vect7);
    T X61M = (*Vect6) - (*Vect1);
    T X25M = (*Vect2) - (*Vect5);
    T X43M = (*Vect4) - (*Vect3);

    T X07P34PP = X07P + X34P;
    T X07P34PM = X07P - X34P;
    T X16P25PP = X16P + X25P;
    T X16P25PM = X16P - X25P;

    (*Vect0) = setval<T>(C_norm) * (X07P34PP + X16P25PP);
    (*Vect2) = setval<T>(C_norm) * (setval<T>(C_b) * X07P34PM + setval<T>(C_e) * X16P25PM);
    (*Vect4) = setval<T>(C_norm) * (X07P34PP - X16P25PP);
    (*Vect6) = setval<T>(C_norm) * (setval<T>(C_e) * X07P34PM - setval<T>(C_b) * X16P25PM);

    (*Vect1) = setval<T>(C_norm) * (setval<T>(C_a) * X07M - setval<T>(C_c) * X61M + setval<T>(C_d) * X25M - setval<T>(C_f) * X43M);
    (*Vect3) = setval<T>(C_norm) * (setval<T>(C_c) * X07M + setval<T>(C_f) * X61M - setval<T>(C_a) * X25M + setval<T>(C_d) * X43M);
    (*Vect5) = setval<T>(C_norm) * (setval<T>(C_d) * X07M + setval<T>(C_a) * X61M + setval<T>(C_f) * X25M - setval<T>(C_c) * X43M);
    (*Vect7) = setval<T>(C_norm) * (setval<T>(C_f) * X07M + setval<T>(C_d) * X61M + setval<T>(C_c) * X25M + setval<T>(C_a) * X43M);
}

template<typename T, int Step>
__device__ void CUDAsubroutineInplaceIDCT8vector(T *Vect0) {
    T *Vect1 = Vect0 + Step;
    T *Vect2 = Vect1 + Step;
    T *Vect3 = Vect2 + Step;
    T *Vect4 = Vect3 + Step;
    T *Vect5 = Vect4 + Step;
    T *Vect6 = Vect5 + Step;
    T *Vect7 = Vect6 + Step;

    T Y04P = (*Vect0) + (*Vect4);
    T Y2b6eP = setval<T>(C_b) * (*Vect2) + setval<T>(C_e) * (*Vect6);

    T Y04P2b6ePP = Y04P + Y2b6eP;
    T Y04P2b6ePM = Y04P - Y2b6eP;
    T Y7f1aP3c5dPP = setval<T>(C_f) * (*Vect7) + setval<T>(C_a) * (*Vect1) + setval<T>(C_c) * (*Vect3) + setval<T>(C_d) * (*Vect5);
    T Y7a1fM3d5cMP = setval<T>(C_a) * (*Vect7) - setval<T>(C_f) * (*Vect1) + setval<T>(C_d) * (*Vect3) - setval<T>(C_c) * (*Vect5);

    T Y04M = (*Vect0) - (*Vect4);
    T Y2e6bM = setval<T>(C_e) * (*Vect2) - setval<T>(C_b) * (*Vect6);

    T Y04M2e6bMP = Y04M + Y2e6bM;
    T Y04M2e6bMM = Y04M - Y2e6bM;
    T Y1c7dM3f5aPM = setval<T>(C_c) * (*Vect1) - setval<T>(C_d) * (*Vect7) - setval<T>(C_f) * (*Vect3) - setval<T>(C_a) * (*Vect5);
    T Y1d7cP3a5fMM = setval<T>(C_d) * (*Vect1) + setval<T>(C_c) * (*Vect7) - setval<T>(C_a) * (*Vect3) + setval<T>(C_f) * (*Vect5);

    (*Vect0) = setval<T>(C_norm) * (Y04P2b6ePP + Y7f1aP3c5dPP);
    (*Vect7) = setval<T>(C_norm) * (Y04P2b6ePP - Y7f1aP3c5dPP);
    (*Vect4) = setval<T>(C_norm) * (Y04P2b6ePM + Y7a1fM3d5cMP);
    (*Vect3) = setval<T>(C_norm) * (Y04P2b6ePM - Y7a1fM3d5cMP);

    (*Vect1) = setval<T>(C_norm) * (Y04M2e6bMP + Y1c7dM3f5aPM);
    (*Vect5) = setval<T>(C_norm) * (Y04M2e6bMM - Y1d7cP3a5fMM);
    (*Vect2) = setval<T>(C_norm) * (Y04M2e6bMM + Y1d7cP3a5fMM);
    (*Vect6) = setval<T>(C_norm) * (Y04M2e6bMP - Y1c7dM3f5aPM);
}

template<typename T, int Step>
__device__ void  CUDAsubroutineInplaceDCT16vector(T *Vect00) {
    T *Vect01 = Vect00 + Step;
    T *Vect02 = Vect01 + Step;
    T *Vect03 = Vect02 + Step;
    T *Vect04 = Vect03 + Step;
    T *Vect05 = Vect04 + Step;
    T *Vect06 = Vect05 + Step;
    T *Vect07 = Vect06 + Step;

    T *Vect08 = Vect00 + (Step << 3);
    T *Vect09 = Vect08 + Step;
    T *Vect10 = Vect09 + Step;
    T *Vect11 = Vect10 + Step;
    T *Vect12 = Vect11 + Step;
    T *Vect13 = Vect12 + Step;
    T *Vect14 = Vect13 + Step;
    T *Vect15 = Vect14 + Step;

    const float x00 = (*Vect00) + (*Vect15);
    const float x01 = (*Vect01) + (*Vect14);
    const float x02 = (*Vect02) + (*Vect13);
    const float x03 = (*Vect03) + (*Vect12);
    const float x04 = (*Vect04) + (*Vect11);
    const float x05 = (*Vect05) + (*Vect10);
    const float x06 = (*Vect06) + (*Vect09);
    const float x07 = (*Vect07) + (*Vect08);
    const float x08 = (*Vect00) - (*Vect15);
    const float x09 = (*Vect01) - (*Vect14);
    const float x0a = (*Vect02) - (*Vect13);
    const float x0b = (*Vect03) - (*Vect12);
    const float x0c = (*Vect04) - (*Vect11);
    const float x0d = (*Vect05) - (*Vect10);
    const float x0e = (*Vect06) - (*Vect09);
    const float x0f = (*Vect07) - (*Vect08);
    const float x10 = x00 + x07;
    const float x11 = x01 + x06;
    const float x12 = x02 + x05;
    const float x13 = x03 + x04;
    const float x14 = x00 - x07;
    const float x15 = x01 - x06;
    const float x16 = x02 - x05;
    const float x17 = x03 - x04;
    const float x18 = x10 + x13;
    const float x19 = x11 + x12;
    const float x1a = x10 - x13;
    const float x1b = x11 - x12;
    const float x1c =   1.38703984532215f*x14 + 0.275899379282943f*x17;
    const float x1d =   1.17587560241936f*x15 + 0.785694958387102f*x16;
    const float x1e = -0.785694958387102f*x15 + 1.17587560241936f *x16;
    const float x1f =  0.275899379282943f*x14 - 1.38703984532215f *x17;
    const float x20 = 0.25f * (x1c - x1d);
    const float x21 = 0.25f * (x1e - x1f);
    const float x22 =  1.40740373752638f *x08 + 0.138617169199091f*x0f;
    const float x23 =  1.35331800117435f *x09 + 0.410524527522357f*x0e;
    const float x24 =  1.24722501298667f *x0a + 0.666655658477747f*x0d;
    const float x25 =  1.09320186700176f *x0b + 0.897167586342636f*x0c;
    const float x26 = -0.897167586342636f*x0b + 1.09320186700176f *x0c;
    const float x27 =  0.666655658477747f*x0a - 1.24722501298667f *x0d;
    const float x28 = -0.410524527522357f*x09 + 1.35331800117435f *x0e;
    const float x29 =  0.138617169199091f*x08 - 1.40740373752638f *x0f;
    const float x2a = x22 + x25;
    const float x2b = x23 + x24;
    const float x2c = x22 - x25;
    const float x2d = x23 - x24;
    const float x2e = 0.25f * (x2a - x2b);
    const float x2f = 0.326640741219094f*x2c + 0.135299025036549f*x2d;
    const float x30 = 0.135299025036549f*x2c - 0.326640741219094f*x2d;
    const float x31 = x26 + x29;
    const float x32 = x27 + x28;
    const float x33 = x26 - x29;
    const float x34 = x27 - x28;
    const float x35 = 0.25f * (x31 - x32);
    const float x36 = 0.326640741219094f*x33 + 0.135299025036549f*x34;
    const float x37 = 0.135299025036549f*x33 - 0.326640741219094f*x34;
    (*Vect00) = 0.25f * (x18 + x19);
    (*Vect01) = 0.25f * (x2a + x2b);
    (*Vect02) = 0.25f * (x1c + x1d);
    (*Vect03) = 0.707106781186547f * (x2f - x37);
    (*Vect04) = 0.326640741219094f * x1a + 0.135299025036549f * x1b;
    (*Vect05) = 0.707106781186547f * (x2f + x37);
    (*Vect06) = 0.707106781186547f * (x20 - x21);
    (*Vect07) = 0.707106781186547f * (x2e + x35);
    (*Vect08) = 0.25f * (x18 - x19);
    (*Vect09) = 0.707106781186547f * (x2e - x35);
    (*Vect10) = 0.707106781186547f * (x20 + x21);
    (*Vect11) = 0.707106781186547f * (x30 - x36);
    (*Vect12) = 0.135299025036549f*x1a - 0.326640741219094f*x1b;
    (*Vect13) = 0.707106781186547f * (x30 + x36);
    (*Vect14) = 0.25f * (x1e + x1f);
    (*Vect15) = 0.25f * (x31 + x32);
}

template<typename T, int Step>
__device__ void  CUDAsubroutineInplaceIDCT16vector(T *Vect00) {
    T *Vect01 = Vect00 + Step;
    T *Vect02 = Vect01 + Step;
    T *Vect03 = Vect02 + Step;
    T *Vect04 = Vect03 + Step;
    T *Vect05 = Vect04 + Step;
    T *Vect06 = Vect05 + Step;
    T *Vect07 = Vect06 + Step;

    T *Vect08 = Vect00 + (Step << 3);
    T *Vect09 = Vect08 + Step;
    T *Vect10 = Vect09 + Step;
    T *Vect11 = Vect10 + Step;
    T *Vect12 = Vect11 + Step;
    T *Vect13 = Vect12 + Step;
    T *Vect14 = Vect13 + Step;
    T *Vect15 = Vect14 + Step;

    const float x00 =  1.4142135623731f   * (*Vect00);
    const float x01 =  1.40740373752638f  * (*Vect01) + 0.138617169199091f * (*Vect15);
    const float x02 =  1.38703984532215f  * (*Vect02) + 0.275899379282943f * (*Vect14);
    const float x03 =  1.35331800117435f  * (*Vect03) + 0.410524527522357f * (*Vect13);
    const float x04 =  1.30656296487638f  * (*Vect04) + 0.541196100146197f * (*Vect12);
    const float x05 =  1.24722501298667f  * (*Vect05) + 0.666655658477747f * (*Vect11);
    const float x06 =  1.17587560241936f  * (*Vect06) + 0.785694958387102f * (*Vect10);
    const float x07 =  1.09320186700176f  * (*Vect07) + 0.897167586342636f * (*Vect09);
    const float x08 =  1.4142135623731f   * (*Vect08);
    const float x09 = -0.897167586342636f * (*Vect07) + 1.09320186700176f * (*Vect09);
    const float x0a =  0.785694958387102f * (*Vect06) - 1.17587560241936f * (*Vect10);
    const float x0b = -0.666655658477747f * (*Vect05) + 1.24722501298667f * (*Vect11);
    const float x0c =  0.541196100146197f * (*Vect04) - 1.30656296487638f * (*Vect12);
    const float x0d = -0.410524527522357f * (*Vect03) + 1.35331800117435f * (*Vect13);
    const float x0e =  0.275899379282943f * (*Vect02) - 1.38703984532215f * (*Vect14);
    const float x0f = -0.138617169199091f * (*Vect01) + 1.40740373752638f * (*Vect15);
    const float x12 = x00 + x08;
    const float x13 = x01 + x07;
    const float x14 = x02 + x06;
    const float x15 = x03 + x05;
    const float x16 = 1.4142135623731f*x04;
    const float x17 = x00 - x08;
    const float x18 = x01 - x07;
    const float x19 = x02 - x06;
    const float x1a = x03 - x05;
    const float x1d = x12 + x16;
    const float x1e = x13 + x15;
    const float x1f = 1.4142135623731f*x14;
    const float x20 = x12 - x16;
    const float x21 = x13 - x15;
    const float x22 = 0.25f * (x1d - x1f);
    const float x23 = 0.25f * (x20 + x21);
    const float x24 = 0.25f * (x20 - x21);
    const float x25 = 1.4142135623731f*x17;
    const float x26 = 1.30656296487638f*x18 + 0.541196100146197f*x1a;
    const float x27 = 1.4142135623731f*x19;
    const float x28 = -0.541196100146197f*x18 + 1.30656296487638f*x1a;
    const float x29 = 0.176776695296637f * (x25 + x27) + 0.25f*x26;
    const float x2a = 0.25f * (x25 - x27);
    const float x2b = 0.176776695296637f * (x25 + x27) - 0.25f*x26;
    const float x2c = 0.353553390593274f*x28;
    const float x1b = 0.707106781186547f * (x2a - x2c);
    const float x1c = 0.707106781186547f * (x2a + x2c);
    const float x2d = 1.4142135623731f*x0c;
    const float x2e = x0b + x0d;
    const float x2f = x0a + x0e;
    const float x30 = x09 + x0f;
    const float x31 = x09 - x0f;
    const float x32 = x0a - x0e;
    const float x33 = x0b - x0d;
    const float x37 = 1.4142135623731f*x2d;
    const float x38 = 1.30656296487638f*x2e + 0.541196100146197f*x30;
    const float x39 = 1.4142135623731f*x2f;
    const float x3a = -0.541196100146197f*x2e + 1.30656296487638f*x30;
    const float x3b = 0.176776695296637f * (x37 + x39) + 0.25f*x38;
    const float x3c = 0.25f * (x37 - x39);
    const float x3d = 0.176776695296637f * (x37 + x39) - 0.25f*x38;
    const float x3e = 0.353553390593274f*x3a;
    const float x34 = 0.707106781186547f * (x3c - x3e);
    const float x35 = 0.707106781186547f * (x3c + x3e);
    const float x3f = 1.4142135623731f*x32;
    const float x40 = x31 + x33;
    const float x41 = x31 - x33;
    const float x42 = 0.25f * (x3f + x40);
    const float x43 = 0.25f * (x3f - x40);
    const float x44 = 0.353553390593274f*x41;
    (*Vect00) = 0.176776695296637f * (x1d + x1f) + 0.25f * x1e;
    (*Vect01) = 0.707106781186547f * (x29 + x3d);
    (*Vect02) = 0.707106781186547f * (x29 - x3d);
    (*Vect03) = 0.707106781186547f * (x23 - x43);
    (*Vect04) = 0.707106781186547f * (x23 + x43);
    (*Vect05) = 0.707106781186547f * (x1b - x35);
    (*Vect06) = 0.707106781186547f * (x1b + x35);
    (*Vect07) = 0.707106781186547f * (x22 + x44);
    (*Vect08) = 0.707106781186547f * (x22 - x44);
    (*Vect09) = 0.707106781186547f * (x1c + x34);
    (*Vect10) = 0.707106781186547f * (x1c - x34);
    (*Vect11) = 0.707106781186547f * (x24 + x42);
    (*Vect12) = 0.707106781186547f * (x24 - x42);
    (*Vect13) = 0.707106781186547f * (x2b - x3b);
    (*Vect14) = 0.707106781186547f * (x2b + x3b);
    (*Vect15) = 0.176776695296637f * (x1d + x1f) - 0.25f*x1e;
}

template<typename T, int BLOCK_SIZE>
__device__ void dctBlock(T shared_tmp[BLOCK_SIZE][BLOCK_SIZE + 1], int thWorker) {
    static_assert(BLOCK_SIZE == 8 || BLOCK_SIZE == 16, "BLOCK_SIZE must be 8 or 16");
    if (BLOCK_SIZE == 8) {
        CUDAsubroutineInplaceDCT8vector<T, 1>             ((T *)&shared_tmp[thWorker][0]); // row
        CUDAsubroutineInplaceDCT8vector<T, BLOCK_SIZE + 1>((T *)&shared_tmp[0][thWorker]); // column
    } else if (BLOCK_SIZE == 16) {
        CUDAsubroutineInplaceDCT16vector<T, 1>             ((T *)&shared_tmp[thWorker][0]); // row
        CUDAsubroutineInplaceDCT16vector<T, BLOCK_SIZE + 1>((T *)&shared_tmp[0][thWorker]); // column
    }
}

template<typename T, int BLOCK_SIZE>
__device__ void idctBlock(T shared_tmp[BLOCK_SIZE][BLOCK_SIZE + 1], int thWorker) {
    static_assert(BLOCK_SIZE == 8 || BLOCK_SIZE == 16, "BLOCK_SIZE must be 8 or 16");
    if (BLOCK_SIZE == 8) {
        CUDAsubroutineInplaceIDCT8vector<T, BLOCK_SIZE+1>((T *)&shared_tmp[0][thWorker]); // column
        CUDAsubroutineInplaceIDCT8vector<T, 1>           ((T *)&shared_tmp[thWorker][0]); // row
    } else if (BLOCK_SIZE == 16) {
        CUDAsubroutineInplaceIDCT16vector<T, BLOCK_SIZE + 1>((T *)&shared_tmp[0][thWorker]); // column
        CUDAsubroutineInplaceIDCT16vector<T, 1>             ((T *)&shared_tmp[thWorker][0]); // row
    }
}

template<typename TypeTmp, int BLOCK_SIZE>
__device__ void thresholdBlock(TypeTmp shared_tmp[BLOCK_SIZE][BLOCK_SIZE + 1], int thWorker, const float threshold) {
    #pragma unroll
    for (int y = 0; y < BLOCK_SIZE; y++) {
        if (y > 0 || thWorker > 0) {
            TypeTmp *ptr = &shared_tmp[y][thWorker];
            const TypeTmp val = ptr[0];
            if (fabs(val) <= threshold) {
                ptr[0] = 0.0f;
            }
        }
    }
}

#define SHARED_TMP TypeTmp shared_tmp[DENOISE_BLOCK_SIZE_X][BLOCK_SIZE][BLOCK_SIZE + 1]
#define SHARED_OUT TypeTmp shared_out[BLOCK_SIZE * DENOISE_SHARED_BLOCK_NUM_Y][BLOCK_SIZE * DENOISE_SHARED_BLOCK_NUM_X]


template<typename TypeTmp, int BLOCK_SIZE>
__device__ void clearSharedOutLine(
    SHARED_OUT,
    const int local_bx,
    const int thWorker,
    const int sy
) {
    const int y = sy % (BLOCK_SIZE * DENOISE_SHARED_BLOCK_NUM_Y);
    shared_out[y][local_bx * BLOCK_SIZE + thWorker] = 0;
    if (local_bx < (DENOISE_SHARED_BLOCK_NUM_X - DENOISE_BLOCK_SIZE_X)) {
        shared_out[y][(local_bx + DENOISE_BLOCK_SIZE_X) * BLOCK_SIZE + thWorker] = 0;
    }
}


template<typename TypeTmp, int BLOCK_SIZE>
__device__ void clearSharedOut(
    SHARED_OUT,
    const int local_bx,
    const int thWorker
) {
    #pragma unroll
    for (int y = 0; y < BLOCK_SIZE * DENOISE_SHARED_BLOCK_NUM_Y; y++) {
        clearSharedOutLine<TypeTmp, BLOCK_SIZE>(shared_out, local_bx, thWorker, y);
    }
}

template<typename TypePixel, typename TypeTmp, int BLOCK_SIZE>
__device__ void loadBlocktmp(
    SHARED_TMP,
    const int local_bx, const int thWorker,
    const char *const __restrict__ ptrSrc, const int srcPitch,
    const int block_x, const int block_y,
    const int width, const int height) {
    #pragma unroll
    for (int y = 0; y < BLOCK_SIZE; y++) {
        const int src_x = wrap_idx(block_x + thWorker, 0, width  - 1);
        const int src_y = wrap_idx(block_y + y,        0, height - 1);
        TypePixel pix = ((const TypePixel *)(ptrSrc + src_y * srcPitch + src_x * sizeof(TypePixel)))[0];
        shared_tmp[local_bx][y][thWorker] = (TypeTmp)pix;
    }
}

template<typename TypeTmp, int BLOCK_SIZE>
__device__ void addBlocktmp(
    SHARED_OUT,
    const int shared_block_x, const int shared_block_y,
    const SHARED_TMP,
    const int local_bx, const int thWorker) {
    #pragma unroll
    for (int y = 0; y < BLOCK_SIZE; y++) {
        shared_out[(shared_block_y + y) % (BLOCK_SIZE * DENOISE_SHARED_BLOCK_NUM_Y)][shared_block_x + thWorker]
            += shared_tmp[local_bx][y][thWorker];
    }
}

// デバッグ用
template<typename TypePixel, typename TypeTmp, int BLOCK_SIZE>
__device__ void directAddBlock(
    SHARED_OUT,
    const int shared_block_x, const int shared_block_y,
    const int thWorker,
    const char *const __restrict__ ptrSrc, const int srcPitch,
    const int block_x, const int block_y,
    const int width, const int height) {
    #pragma unroll
    for (int y = 0; y < BLOCK_SIZE; y++) {
        const int src_x = wrap_idx(block_x + thWorker, 0, width - 1);
        const int src_y = wrap_idx(block_y + y,        0, height - 1);
        TypePixel pix = ((const TypePixel *)(ptrSrc + src_y * srcPitch + src_x * sizeof(TypePixel)))[0];
        shared_out[(shared_block_y + y) % (BLOCK_SIZE * DENOISE_SHARED_BLOCK_NUM_Y)][shared_block_x + thWorker] += pix;
    }
}

template<typename TypePixel, int bit_depth, typename TypeTmp, int BLOCK_SIZE>
__device__ void filter_block(
    const char *const __restrict__ ptrSrc, const int srcPitch,
    SHARED_TMP,
    SHARED_OUT,
    const int local_bx, const int thWorker,
    const int shared_block_x, const int shared_block_y,
    const int block_x, const int block_y,
    const int width, const int height,
    const float threshold) {
#if 1
    loadBlocktmp<TypePixel, TypeTmp, BLOCK_SIZE>(shared_tmp, local_bx, thWorker, ptrSrc, srcPitch, block_x, block_y, width, height);
    dctBlock<TypeTmp, BLOCK_SIZE>(shared_tmp[local_bx], thWorker);
    thresholdBlock<TypeTmp, BLOCK_SIZE>(shared_tmp[local_bx], thWorker, threshold);
    idctBlock<TypeTmp, BLOCK_SIZE>(shared_tmp[local_bx], thWorker);
    addBlocktmp<TypeTmp, BLOCK_SIZE>(shared_out, shared_block_x, shared_block_y, shared_tmp, local_bx, thWorker);
#else
    directAddBlock<TypePixel, TypeTmp, BLOCK_SIZE>(shared_out, shared_block_x, shared_block_y, thWorker, ptrSrc, srcPitch, block_x, block_y, width, height);
#endif
}

template<typename TypePixel, int bit_depth, typename TypeTmp, typename TypeWeight, int BLOCK_SIZE, int STEP>
__device__ void write_output(
    char *const __restrict__ ptrDst, const int dstPitch,
    SHARED_OUT,
    const int width, const int height,
    const int sx, const int sy, 
    const int x, const int y) {
    if (x < width && y < height) {
        TypePixel*dst = (TypePixel*)(ptrDst + y * dstPitch + x * sizeof(TypePixel));
        const TypeTmp *out = &shared_out[sy % (BLOCK_SIZE * DENOISE_SHARED_BLOCK_NUM_Y)][sx];
        const float weight = (1.0f / (float)(BLOCK_SIZE * BLOCK_SIZE / (STEP * STEP)));
        dst[0] = out[0] * weight;
    }
}

template<typename TypePixel, int bit_depth, typename TypeTmp, typename TypeWeight, int BLOCK_SIZE, int STEP>
__global__ void kernel_denoise_dct(
    char *const __restrict__ ptrDst0,
    char *const __restrict__ ptrDst1,
    char *const __restrict__ ptrDst2,
    const int dstPitch,
    const char *const __restrict__ ptrSrc0,
    const char *const __restrict__ ptrSrc1,
    const char *const __restrict__ ptrSrc2,
    const int srcPitch,
    const int width, const int height,
    const float threshold) {
    const int thWorker = threadIdx.x; // BLOCK_SIZE
    const int local_bx = threadIdx.y; // DENOISE_BLOCK_SIZE_X
    const int global_bx = blockIdx.x * DENOISE_BLOCK_SIZE_X + local_bx;
    const int global_by = blockIdx.y * DENOISE_LOOP_COUNT_BLOCK;
    const int plane_idx = blockIdx.z;

    const int block_x = global_bx * BLOCK_SIZE;
    const int block_y = global_by * BLOCK_SIZE;

    char *const __restrict__ ptrDst = selectptr(ptrDst0, ptrDst1, ptrDst2, plane_idx);
    const char *const __restrict__ ptrSrc = selectptr(ptrSrc0, ptrSrc1, ptrSrc2, plane_idx);

    __shared__ SHARED_TMP;
    __shared__ SHARED_OUT;

    #define FILTER_BLOCK(SHARED_X, SHARED_Y, X, Y) \
        { filter_block<TypePixel, bit_depth, TypeTmp, BLOCK_SIZE>(ptrSrc, srcPitch, shared_tmp, shared_out, local_bx, thWorker, (SHARED_X), (SHARED_Y), (X), (Y), width, height, threshold); }

    { // SHARED_OUTの初期化
        clearSharedOut<TypeTmp, BLOCK_SIZE>(shared_out, local_bx, thWorker);
        __syncthreads();
    }

    { // y方向の事前計算
        const int block_y_start = (block_y - BLOCK_SIZE) + STEP;
        for (int y = block_y_start; y < block_y; y += STEP) {
            const int shared_y = y - (block_y - BLOCK_SIZE);
            for (int ix_loop = 0; ix_loop < BLOCK_SIZE; ix_loop += STEP) {
                const int x = block_x + ix_loop;
                const int shared_x = local_bx * BLOCK_SIZE + ix_loop;
                if (local_bx < 1) { // x方向の事前計算
                    FILTER_BLOCK(shared_x, shared_y, x - BLOCK_SIZE, y);
                }
                FILTER_BLOCK(shared_x + BLOCK_SIZE, shared_y, x, y);
                __syncthreads();
            }
        }
    }

    { // 本計算
        const int block_y_fin = min(height, block_y + DENOISE_LOOP_COUNT_BLOCK * BLOCK_SIZE);
        for (int y = block_y; y < block_y_fin; y += STEP) {
            const int shared_y = y - (block_y - BLOCK_SIZE);
            for (int ix_loop = 0; ix_loop < BLOCK_SIZE; ix_loop += STEP) {
                const int x = block_x + ix_loop;
                const int shared_x = local_bx * BLOCK_SIZE + ix_loop;
                if (local_bx < 1) { // x方向の事前計算
                    FILTER_BLOCK(shared_x, shared_y, x - BLOCK_SIZE, y);
                }
                FILTER_BLOCK(shared_x + BLOCK_SIZE, shared_y, x, y);
                __syncthreads();
            }
            for (int iy = 0; iy < STEP; iy++) {
                write_output<TypePixel, bit_depth, TypeTmp, TypeWeight, BLOCK_SIZE, STEP>(ptrDst, dstPitch, shared_out, width, height,
                    (local_bx + 1 /*1ブロック分ずれている*/) * BLOCK_SIZE + thWorker, shared_y + iy, block_x + thWorker, y + iy);

                clearSharedOutLine<TypeTmp, BLOCK_SIZE>(shared_out, local_bx, thWorker, shared_y + iy + BLOCK_SIZE /*1ブロック先をクリア*/);
            }
            __syncthreads();
        }
    }
    #undef FILTER_BLOCK
}


template<typename Type, int bit_depth, int BLOCK_SIZE, int STEP>
RGY_ERR denoise_dct_run(RGYFrameInfo *pOutputFrame, const RGYFrameInfo *pInputFrame,
    const float threshold, hipStream_t stream) {
    const auto planeInputR = getPlane(pInputFrame, RGY_PLANE_R);
    const auto planeInputG = getPlane(pInputFrame, RGY_PLANE_G);
    const auto planeInputB = getPlane(pInputFrame, RGY_PLANE_B);
    auto planeOutputR = getPlane(pOutputFrame, RGY_PLANE_R);
    auto planeOutputG = getPlane(pOutputFrame, RGY_PLANE_G);
    auto planeOutputB = getPlane(pOutputFrame, RGY_PLANE_B);
    if (planeInputR.pitchArray[0] != planeInputG.pitchArray[0] || planeInputR.pitchArray[0] != planeInputB.pitchArray[0]
        || planeOutputR.pitchArray[0] != planeOutputG.pitchArray[0] || planeOutputR.pitchArray[0] != planeOutputB.pitchArray[0]) {
        return RGY_ERR_UNKNOWN;
    }
    dim3 blockSize(BLOCK_SIZE, DENOISE_BLOCK_SIZE_X);
    dim3 gridSize(divCeil(planeInputR.width, blockSize.x * DENOISE_BLOCK_SIZE_X), divCeil(planeInputR.height, BLOCK_SIZE * DENOISE_LOOP_COUNT_BLOCK), 3);
    kernel_denoise_dct<Type, bit_depth, float, float, BLOCK_SIZE, STEP> << <gridSize, blockSize, 0, stream >>>(
        (char *)planeOutputR.ptrArray[0], (char *)planeOutputG.ptrArray[0], (char *)planeOutputB.ptrArray[0], planeOutputR.pitchArray[0],
        (const char *)planeInputR.ptrArray[0], (const char *)planeInputG.ptrArray[0], (const char *)planeInputB.ptrArray[0], planeInputR.pitchArray[0],
        planeInputR.width, planeInputR.height, threshold);
    auto err = err_to_rgy(hipGetLastError());
    if (err != RGY_ERR_NONE) {
        return err;
    }
    return err;
}

template<typename Type, int bit_depth, int BLOCK_SIZE>
static RGY_ERR denoise_frame(RGYFrameInfo *pOutputFrame, const RGYFrameInfo *pInputFrame,
    const float threshold, const int step, hipStream_t stream) {
    switch (step) {
    case 2:  return denoise_dct_run<Type, bit_depth, BLOCK_SIZE, 2>(pOutputFrame, pInputFrame, threshold, stream);
    case 4:  return denoise_dct_run<Type, bit_depth, BLOCK_SIZE, 4>(pOutputFrame, pInputFrame, threshold, stream);
    case 8:  return denoise_dct_run<Type, bit_depth, BLOCK_SIZE, 8>(pOutputFrame, pInputFrame, threshold, stream);
    default: return denoise_dct_run<Type, bit_depth, BLOCK_SIZE, 1>(pOutputFrame, pInputFrame, threshold, stream);
    }
}

template<typename Type>
__global__ void kernel_color_decorrelation(
    uint8_t *__restrict__ dst0, uint8_t *__restrict__ dst1, uint8_t *__restrict__ dst2, const int dstPitch,
    const uint8_t *__restrict__ src0, const uint8_t *__restrict__ src1, const uint8_t *__restrict__ src2, const int srcPitch,
    const int width, const int height) {
    const int ix = blockIdx.x * blockDim.x + threadIdx.x;
    const int iy = blockIdx.y * blockDim.y + threadIdx.y;
    if (ix < width && iy < height) {
        const float ptrSrc0 = (float)(((const Type *)(src0 + iy * srcPitch + ix * sizeof(Type)))[0]);
        const float ptrSrc1 = (float)(((const Type *)(src1 + iy * srcPitch + ix * sizeof(Type)))[0]);
        const float ptrSrc2 = (float)(((const Type *)(src2 + iy * srcPitch + ix * sizeof(Type)))[0]);

        const float d0 = ptrSrc0 * DCT3X3_0_0 + ptrSrc1 * DCT3X3_0_1 + ptrSrc2 * DCT3X3_0_2;
        const float d1 = ptrSrc0 * DCT3X3_1_0 +                        ptrSrc2 * DCT3X3_1_2;
        const float d2 = ptrSrc0 * DCT3X3_2_0 + ptrSrc1 * DCT3X3_2_1 + ptrSrc2 * DCT3X3_2_2;

        Type *ptrDst0 = (Type *)(dst0 + iy * dstPitch + ix * sizeof(Type));
        Type *ptrDst1 = (Type *)(dst1 + iy * dstPitch + ix * sizeof(Type));
        Type *ptrDst2 = (Type *)(dst2 + iy * dstPitch + ix * sizeof(Type));
        ptrDst0[0] = d0;
        ptrDst1[0] = d1;
        ptrDst2[0] = d2;
    }
}

RGY_ERR NVEncFilterDenoiseDct::colorDecorrelation(RGYFrameInfo *pOutputFrame, const RGYFrameInfo *pInputFrame, hipStream_t stream) {
    const auto planeInputR = getPlane(pInputFrame, RGY_PLANE_R);
    const auto planeInputG = getPlane(pInputFrame, RGY_PLANE_G);
    const auto planeInputB = getPlane(pInputFrame, RGY_PLANE_B);
    auto planeOutputR = getPlane(pOutputFrame, RGY_PLANE_R);
    auto planeOutputG = getPlane(pOutputFrame, RGY_PLANE_G);
    auto planeOutputB = getPlane(pOutputFrame, RGY_PLANE_B);
    if (   cmpFrameInfoCspResolution(&planeInputR, &planeOutputR)
        || cmpFrameInfoCspResolution(&planeInputG, &planeOutputG)
        || cmpFrameInfoCspResolution(&planeInputB, &planeOutputB)
        || cmpFrameInfoCspResolution(&planeInputR, &planeInputG)
        || cmpFrameInfoCspResolution(&planeInputR, &planeInputB)) {
        return RGY_ERR_UNKNOWN;
    }
    if (planeInputR.pitchArray[0] != planeInputG.pitchArray[0] || planeInputR.pitchArray[0] != planeInputB.pitchArray[0]
        || planeOutputR.pitchArray[0] != planeOutputG.pitchArray[0] || planeOutputR.pitchArray[0] != planeOutputB.pitchArray[0]) {
        return RGY_ERR_UNKNOWN;
    }
    dim3 blockSize(64, 8);
    dim3 gridSize(divCeil(planeInputR.width, blockSize.x), divCeil(planeInputR.height, blockSize.y));
    kernel_color_decorrelation<float> << <gridSize, blockSize, 0, stream >> > (
        planeOutputR.ptrArray[0], planeOutputG.ptrArray[0], planeOutputB.ptrArray[0], planeOutputR.pitchArray[0],
        planeInputR.ptrArray[0], planeInputG.ptrArray[0], planeInputB.ptrArray[0], planeInputR.pitchArray[0],
        planeInputR.width, planeInputR.height);
    auto err = err_to_rgy(hipGetLastError());
    if (err != RGY_ERR_NONE) {
        return err;
    }
    return err;
}


template<typename Type>
__global__ void kernel_color_correlation(
    uint8_t *__restrict__ dst0, uint8_t *__restrict__ dst1, uint8_t *__restrict__ dst2, const int dstPitch,
    const uint8_t *__restrict__ src0, const uint8_t *__restrict__ src1, const uint8_t *__restrict__ src2, const int srcPitch,
    const int width, const int height) {
    const int ix = blockIdx.x * blockDim.x + threadIdx.x;
    const int iy = blockIdx.y * blockDim.y + threadIdx.y;
    if (ix < width && iy < height) {
        const float ptrSrc0 = (float)(((const Type *)(src0 + iy * srcPitch + ix * sizeof(Type)))[0]);
        const float ptrSrc1 = (float)(((const Type *)(src1 + iy * srcPitch + ix * sizeof(Type)))[0]);
        const float ptrSrc2 = (float)(((const Type *)(src2 + iy * srcPitch + ix * sizeof(Type)))[0]);

        const float d0 = ptrSrc0 * DCT3X3_0_0 + ptrSrc1 * DCT3X3_1_0 + ptrSrc2 * DCT3X3_2_0;
        const float d1 = ptrSrc0 * DCT3X3_0_1                        + ptrSrc2 * DCT3X3_2_1;
        const float d2 = ptrSrc0 * DCT3X3_0_2 + ptrSrc1 * DCT3X3_1_2 + ptrSrc2 * DCT3X3_2_2;

        Type *ptrDst0 = (Type *)(dst0 + iy * dstPitch + ix * sizeof(Type));
        Type *ptrDst1 = (Type *)(dst1 + iy * dstPitch + ix * sizeof(Type));
        Type *ptrDst2 = (Type *)(dst2 + iy * dstPitch + ix * sizeof(Type));
        ptrDst0[0] = d0;
        ptrDst1[0] = d1;
        ptrDst2[0] = d2;
    }
}

RGY_ERR NVEncFilterDenoiseDct::colorCorrelation(RGYFrameInfo *pOutputFrame, const RGYFrameInfo *pInputFrame, hipStream_t stream) {
    const auto planeInputR = getPlane(pInputFrame, RGY_PLANE_R);
    const auto planeInputG = getPlane(pInputFrame, RGY_PLANE_G);
    const auto planeInputB = getPlane(pInputFrame, RGY_PLANE_B);
    auto planeOutputR = getPlane(pOutputFrame, RGY_PLANE_R);
    auto planeOutputG = getPlane(pOutputFrame, RGY_PLANE_G);
    auto planeOutputB = getPlane(pOutputFrame, RGY_PLANE_B);
    if (   cmpFrameInfoCspResolution(&planeInputR, &planeOutputR)
        || cmpFrameInfoCspResolution(&planeInputG, &planeOutputG)
        || cmpFrameInfoCspResolution(&planeInputB, &planeOutputB)
        || cmpFrameInfoCspResolution(&planeInputR, &planeInputG)
        || cmpFrameInfoCspResolution(&planeInputR, &planeInputB)) {
        return RGY_ERR_UNKNOWN;
    }
    if (planeInputR.pitchArray[0] != planeInputG.pitchArray[0] || planeInputR.pitchArray[0] != planeInputB.pitchArray[0]
        || planeOutputR.pitchArray[0] != planeOutputG.pitchArray[0] || planeOutputR.pitchArray[0] != planeOutputB.pitchArray[0]) {
        return RGY_ERR_UNKNOWN;
    }
    dim3 blockSize(64, 8);
    dim3 gridSize(divCeil(planeInputR.width, blockSize.x), divCeil(planeInputR.height, blockSize.y));
    kernel_color_correlation<float><<<gridSize, blockSize, 0, stream >>> (
        planeOutputR.ptrArray[0], planeOutputG.ptrArray[0], planeOutputB.ptrArray[0], planeOutputR.pitchArray[0],
        planeInputR.ptrArray[0], planeInputG.ptrArray[0], planeInputB.ptrArray[0], planeInputR.pitchArray[0],
        planeInputR.width, planeInputR.height);
    auto err = err_to_rgy(hipGetLastError());
    if (err != RGY_ERR_NONE) {
        return err;
    }
    return err;
}

RGY_ERR NVEncFilterDenoiseDct::denoise(RGYFrameInfo *pOutputFrame, const RGYFrameInfo *pInputFrame, hipStream_t stream) {
    if (m_param->frameOut.csp != m_param->frameIn.csp) {
        AddMessage(RGY_LOG_ERROR, _T("csp does not match.\n"));
        return RGY_ERR_INVALID_PARAM;
    }
    auto prm = std::dynamic_pointer_cast<NVEncFilterParamDenoiseDct>(m_param);
    if (!prm) {
        AddMessage(RGY_LOG_ERROR, _T("Invalid parameter type.\n"));
        return RGY_ERR_INVALID_PARAM;
    }
    CUFrameBuf *bufDst = m_bufImg[0].get();
    {
        RGYFrameInfo srcImgInfo = m_srcCrop->GetFilterParam()->frameOut;
        int cropFilterOutputNum = 0;
        RGYFrameInfo *outInfo[1] = { &bufDst->frame };
        RGYFrameInfo cropInput = *pInputFrame;
        auto sts_filter = m_srcCrop->filter(&cropInput, (RGYFrameInfo **)&outInfo, &cropFilterOutputNum, stream);
        if (outInfo[0] == nullptr || cropFilterOutputNum != 1) {
            AddMessage(RGY_LOG_ERROR, _T("Unknown behavior \"%s\".\n"), m_srcCrop->name().c_str());
            return sts_filter;
        }
        if (sts_filter != RGY_ERR_NONE || cropFilterOutputNum != 1) {
            AddMessage(RGY_LOG_ERROR, _T("Error while running filter \"%s\".\n"), m_srcCrop->name().c_str());
            return sts_filter;
        }
    }
    CUFrameBuf *bufSrc = bufDst;
    bufDst = m_bufImg[1].get();
    auto sts = colorDecorrelation(&bufDst->frame, &bufSrc->frame, stream);
    if (sts != RGY_ERR_NONE) {
        return sts;
    }
#if 1
    std::swap(bufSrc, bufDst);
    static const std::map<int, decltype(denoise_frame<float, 32, 8>)*> func_list = {
        { 8,  denoise_frame<float, 32,  8> },
        { 16, denoise_frame<float, 32, 16> },
    };
    if (func_list.count(prm->dct.block_size) == 0) {
        AddMessage(RGY_LOG_ERROR, _T("unsupported block_size %d.\n"), prm->dct.block_size);
        return RGY_ERR_UNSUPPORTED;
    }
    sts = func_list.at(prm->dct.block_size)(&bufDst->frame, &bufSrc->frame, m_threshold, m_step, stream);
    if (sts != RGY_ERR_NONE) {
        return sts;
    }
#endif
    std::swap(bufSrc, bufDst);
    sts = colorCorrelation(&bufDst->frame, &bufSrc->frame, stream);
    if (sts != RGY_ERR_NONE) {
        return sts;
    }
    std::swap(bufSrc, bufDst);
    {
        int cropFilterOutputNum = 0;
        RGYFrameInfo *outInfo[1] = { pOutputFrame };
        auto sts_filter = m_dstCrop->filter(&bufSrc->frame, outInfo, &cropFilterOutputNum, stream);
        if (outInfo[0] == nullptr || cropFilterOutputNum != 1) {
            AddMessage(RGY_LOG_ERROR, _T("Unknown behavior \"%s\".\n"), m_dstCrop->name().c_str());
            return sts_filter;
        }
    }
    return RGY_ERR_NONE;
}

NVEncFilterDenoiseDct::NVEncFilterDenoiseDct() :
    m_bInterlacedWarn(false),
    m_threshold(0.0f),
    m_step(0),
    m_srcCrop(),
    m_dstCrop(),
    m_bufImg() {
    m_name = _T("denoise-dct");
}

NVEncFilterDenoiseDct::~NVEncFilterDenoiseDct() {
    close();
}

RGY_ERR NVEncFilterDenoiseDct::checkParam(const NVEncFilterParamDenoiseDct *prm) {
    //パラメータチェック
    if (prm->frameOut.height <= 0 || prm->frameOut.width <= 0) {
        AddMessage(RGY_LOG_ERROR, _T("Invalid parameter.\n"));
        return RGY_ERR_INVALID_PARAM;
    }
    if (prm->dct.sigma < 0.0f) {
        AddMessage(RGY_LOG_ERROR, _T("Invalid parameter, sigma must be a positive value.\n"));
        return RGY_ERR_INVALID_PARAM;
    }
    if (get_cx_index(list_vpp_denoise_dct_block_size, prm->dct.block_size) < 0) {
        AddMessage(RGY_LOG_ERROR, _T("Invalid block_size.\n"));
        return RGY_ERR_INVALID_PARAM;
    }
    return RGY_ERR_NONE;
}

RGY_ERR NVEncFilterDenoiseDct::init(shared_ptr<NVEncFilterParam> pParam, shared_ptr<RGYLog> pPrintMes) {
    RGY_ERR sts = RGY_ERR_NONE;
    m_pLog = pPrintMes;
    auto prm = std::dynamic_pointer_cast<NVEncFilterParamDenoiseDct>(pParam);
    if (!prm) {
        AddMessage(RGY_LOG_ERROR, _T("Invalid parameter type.\n"));
        return RGY_ERR_INVALID_PARAM;
    }
    if ((sts = checkParam(prm.get())) != RGY_ERR_NONE) {
        return sts;
    }
    if (!m_param || m_param != pParam) {
        {
            AddMessage(RGY_LOG_DEBUG, _T("Create input csp conversion filter.\n"));
            unique_ptr<NVEncFilterCspCrop> filter(new NVEncFilterCspCrop());
            shared_ptr<NVEncFilterParamCrop> paramCrop(new NVEncFilterParamCrop());
            paramCrop->frameIn = pParam->frameIn;
            paramCrop->frameOut = paramCrop->frameIn;
            paramCrop->frameOut.csp = RGY_CSP_RGB_F32;
            paramCrop->baseFps = pParam->baseFps;
            paramCrop->frameIn.mem_type = RGY_MEM_TYPE_GPU;
            paramCrop->frameOut.mem_type = RGY_MEM_TYPE_GPU;
            paramCrop->bOutOverwrite = false;
            sts = filter->init(paramCrop, m_pLog);
            if (sts != RGY_ERR_NONE) {
                return sts;
            }
            m_srcCrop = std::move(filter);
            AddMessage(RGY_LOG_DEBUG, _T("created %s.\n"), m_srcCrop->GetInputMessage().c_str());
        }
        {
            AddMessage(RGY_LOG_DEBUG, _T("Create output csp conversion filter.\n"));
            unique_ptr<NVEncFilterCspCrop> filter(new NVEncFilterCspCrop());
            shared_ptr<NVEncFilterParamCrop> paramCrop(new NVEncFilterParamCrop());
            paramCrop->frameIn = m_srcCrop->GetFilterParam()->frameOut;
            paramCrop->frameOut = pParam->frameOut;
            paramCrop->baseFps = pParam->baseFps;
            paramCrop->frameIn.mem_type = RGY_MEM_TYPE_GPU;
            paramCrop->frameOut.mem_type = RGY_MEM_TYPE_GPU;
            paramCrop->bOutOverwrite = false;
            sts = filter->init(paramCrop, m_pLog);
            if (sts != RGY_ERR_NONE) {
                return sts;
            }
            m_dstCrop = std::move(filter);
            AddMessage(RGY_LOG_DEBUG, _T("created %s.\n"), m_dstCrop->GetInputMessage().c_str());
        }
        for (auto& buf : m_bufImg) {
            if (!buf || cmpFrameInfoCspResolution(&buf->frame, &m_srcCrop->GetFilterParam()->frameOut)) {
                buf = std::make_unique<CUFrameBuf>(m_srcCrop->GetFilterParam()->frameOut);
                if ((sts = buf->alloc()) != RGY_ERR_NONE) {
                    return sts;
                }
            }
        }

        sts = AllocFrameBuf(prm->frameOut, 1);
        if (sts != RGY_ERR_NONE) {
            AddMessage(RGY_LOG_ERROR, _T("failed to allocate memory: %s.\n"), get_err_mes(sts));
            return sts;
        }
        for (int i = 0; i < RGY_CSP_PLANES[pParam->frameOut.csp]; i++) {
            prm->frameOut.pitchArray[i] = m_frameBuf[0]->frame.pitchArray[i];
        }

        m_step = prm->dct.step;
        m_threshold = prm->dct.sigma * 3.0f / 255.0f;
    }

    setFilterInfo(pParam->print());
    m_param = pParam;
    return sts;
}

tstring NVEncFilterParamDenoiseDct::print() const {
    return dct.print();
}

RGY_ERR NVEncFilterDenoiseDct::run_filter(const RGYFrameInfo *pInputFrame, RGYFrameInfo **ppOutputFrames, int *pOutputFrameNum, hipStream_t stream) {
    RGY_ERR sts = RGY_ERR_NONE;
    if (pInputFrame->ptrArray[0] == nullptr) {
        return sts;
    }

    *pOutputFrameNum = 1;
    if (ppOutputFrames[0] == nullptr) {
        auto pOutFrame = m_frameBuf[m_nFrameIdx].get();
        ppOutputFrames[0] = &pOutFrame->frame;
        m_nFrameIdx = (m_nFrameIdx + 1) % m_frameBuf.size();
    }
    ppOutputFrames[0]->picstruct = pInputFrame->picstruct;
    //if (interlaced(*pInputFrame)) {
    //    return filter_as_interlaced_pair(pInputFrame, ppOutputFrames[0], hipStreamDefault);
    //}
    const auto memcpyKind = getCudaMemcpyKind(pInputFrame->mem_type, ppOutputFrames[0]->mem_type);
    if (memcpyKind != hipMemcpyDeviceToDevice) {
        AddMessage(RGY_LOG_ERROR, _T("only supported on device memory.\n"));
        return RGY_ERR_INVALID_PARAM;
    }
    if (m_param->frameOut.csp != m_param->frameIn.csp) {
        AddMessage(RGY_LOG_ERROR, _T("csp does not match.\n"));
        return RGY_ERR_INVALID_PARAM;
    }
    sts = denoise(ppOutputFrames[0], pInputFrame, stream);
    if (sts != RGY_ERR_NONE) {
        AddMessage(RGY_LOG_ERROR, _T("error at denoise: %s.\n"),
            RGY_CSP_NAMES[pInputFrame->csp],
            get_err_mes(sts));
        return sts;
    }
    return sts;
}

void NVEncFilterDenoiseDct::close() {
    m_frameBuf.clear();
    m_bInterlacedWarn = false;
}
