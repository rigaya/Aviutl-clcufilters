#include "hip/hip_runtime.h"
﻿// -----------------------------------------------------------------------------------------
// NVEnc by rigaya
// -----------------------------------------------------------------------------------------
//
// The MIT License
//
// Copyright (c) 2014-2016 rigaya
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
// THE SOFTWARE.
//
// ------------------------------------------------------------------------------------------

#include <map>
#include <array>
#include "convert_csp.h"
#include "NVEncFilterDenoiseKnn.h"
#include "rgy_prm.h"
#pragma warning (push)
#pragma warning (disable: 4819)
#include "hip/hip_runtime.h"
#include ""
#pragma warning (pop)
#include "rgy_cuda_util_kernel.h"

static const int KNN_RADIUS_MAX = 5;

template<typename Type, int knn_radius, int bit_depth>
__global__ void kernel_denoise_knn(uint8_t *__restrict__ pDst, const int dstPitch, const int dstWidth, const int dstHeight,
    hipTextureObject_t texSrc, const float strength, const float lerpC, const float weight_threshold, const float lerp_threshold) {
    const float knn_window_area = (float)((2 * knn_radius + 1) * (2 * knn_radius + 1));
    const float inv_knn_window_area = 1.0f / knn_window_area;
    const int ix = blockIdx.x * blockDim.x + threadIdx.x;
    const int iy = blockIdx.y * blockDim.y + threadIdx.y;
    if (ix < dstWidth && iy < dstHeight) {
        const float x = (float)ix + 0.5f;
        const float y = (float)iy + 0.5f;

        float fCount = 0.0f;
        float sumWeights = 0.0f;
        float sum = 0.0f;
        float center = (float)tex2D<Type>(texSrc, x, y) * (1.0f / (1<<bit_depth));

        #pragma unroll
        for (int i = -knn_radius; i <= knn_radius; i++) {
            #pragma unroll
            for (int j = -knn_radius; j <= knn_radius; j++) {
                float clrIJ = (float)tex2D<Type>(texSrc, x + (float)j, y + (float)i) * (1.0f / (1<<bit_depth));
                float distanceIJ = (center - clrIJ) * (center - clrIJ);

                float weightIJ = __expf(-(distanceIJ * strength + (float)(i * i + j * j) * inv_knn_window_area));

                sum += clrIJ * weightIJ;

                sumWeights += weightIJ;

                fCount += (weightIJ > weight_threshold) ? inv_knn_window_area : 0;
            }
        }
        float lerpQ = (fCount > lerp_threshold) ? lerpC : 1.0f - lerpC;

        Type *ptr = (Type *)(pDst + iy * dstPitch + ix * sizeof(Type));
        ptr[0] = (Type)(lerpf(sum * __frcp_rn(sumWeights), center, lerpQ) * (1<<bit_depth));
    }
}

template<typename Type, int bit_depth>
void denoise_knn(uint8_t *pDst, const int dstPitch, const int dstWidth, const int dstHeight,
    hipTextureObject_t texSrc, int radius, const float strength, const float lerpC, const float weight_threshold, const float lerp_threshold,
    hipStream_t stream) {
    dim3 blockSize(64, 16);
    dim3 gridSize(divCeil(dstWidth, blockSize.x), divCeil(dstHeight, blockSize.y));
    switch (radius) {
    case 1:
        kernel_denoise_knn<Type, 1, bit_depth><<<gridSize, blockSize, 0, stream>>>(pDst, dstPitch, dstWidth, dstHeight, texSrc,
            1.0f / (strength * strength), lerpC, weight_threshold, lerp_threshold);
        break;
    case 2:
        kernel_denoise_knn<Type, 2, bit_depth><<<gridSize, blockSize, 0, stream>>>(pDst, dstPitch, dstWidth, dstHeight, texSrc,
            1.0f / (strength * strength), lerpC, weight_threshold, lerp_threshold);
        break;
    case 3:
        kernel_denoise_knn<Type, 3, bit_depth><<<gridSize, blockSize, 0, stream>>>(pDst, dstPitch, dstWidth, dstHeight, texSrc,
            1.0f / (strength * strength), lerpC, weight_threshold, lerp_threshold);
        break;
    case 4:
        kernel_denoise_knn<Type, 4, bit_depth><<<gridSize, blockSize, 0, stream>>>(pDst, dstPitch, dstWidth, dstHeight, texSrc,
            1.0f / (strength * strength), lerpC, weight_threshold, lerp_threshold);
        break;
    case 5:
        //よりレジスタを使うので、ブロック当たりのスレッド数を低減
        blockSize = dim3(32, 16);
        gridSize = dim3(divCeil(dstWidth, blockSize.x), divCeil(dstHeight, blockSize.y));
        kernel_denoise_knn<Type, 5, bit_depth><<<gridSize, blockSize, 0, stream>>>(pDst, dstPitch, dstWidth, dstHeight, texSrc,
            1.0f / (strength * strength), lerpC, weight_threshold, lerp_threshold);
        break;
    default:
        break;
    }
}

template<typename Type>
hipError_t textureCreateDenoiseKnn(hipTextureObject_t& tex, hipTextureFilterMode filterMode, hipTextureReadMode readMode, uint8_t *ptr, int pitch, int width, int height) {
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypePitch2D;
    resDesc.res.pitch2D.devPtr = ptr;
    resDesc.res.pitch2D.pitchInBytes = pitch;
    resDesc.res.pitch2D.width = width;
    resDesc.res.pitch2D.height = height;
    resDesc.res.pitch2D.desc = hipCreateChannelDesc<Type>();

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0]   = hipAddressModeClamp;
    texDesc.addressMode[1]   = hipAddressModeClamp;
    texDesc.filterMode       = filterMode;
    texDesc.readMode         = readMode;
    texDesc.normalizedCoords = 0;

    return hipCreateTextureObject(&tex, &resDesc, &texDesc, nullptr);
}

template<typename Type, int bit_depth>
static hipError_t denoise_knn_plane(RGYFrameInfo *pOutputFrame, const RGYFrameInfo *pInputFrame,
    int radius, const float strength, const float lerpC, const float weight_threshold, const float lerp_threshold,
    hipStream_t stream) {
    hipTextureObject_t texSrc = 0;
    auto cudaerr = textureCreateDenoiseKnn<Type>(texSrc, hipFilterModePoint, hipReadModeElementType, pInputFrame->ptr[0], pInputFrame->pitch[0], pInputFrame->width, pInputFrame->height);
    if (cudaerr != hipSuccess) {
        return cudaerr;
    }
    denoise_knn<Type, bit_depth>((uint8_t *)pOutputFrame->ptr[0],
        pOutputFrame->pitch[0], pOutputFrame->width, pOutputFrame->height,
        texSrc, radius, strength, lerpC, weight_threshold, lerp_threshold, stream);
    cudaerr = hipGetLastError();
    if (cudaerr != hipSuccess) {
        return cudaerr;
    }
    cudaerr = hipDestroyTextureObject(texSrc);
    if (cudaerr != hipSuccess) {
        return cudaerr;
    }
    return cudaerr;
}

template<typename Type, int bit_depth>
static hipError_t denoise_knn_frame(RGYFrameInfo *pOutputFrame, const RGYFrameInfo *pInputFrame,
    int radius, const float strength, const float lerpC, const float weight_threshold, const float lerp_threshold,
    hipStream_t stream) {
    hipError_t cudaerr = hipSuccess;
    const auto planeInputY = getPlane(pInputFrame, RGY_PLANE_Y);
    const auto planeInputU = getPlane(pInputFrame, RGY_PLANE_U);
    const auto planeInputV = getPlane(pInputFrame, RGY_PLANE_V);
    auto planeOutputY = getPlane(pOutputFrame, RGY_PLANE_Y);
    auto planeOutputU = getPlane(pOutputFrame, RGY_PLANE_U);
    auto planeOutputV = getPlane(pOutputFrame, RGY_PLANE_V);

    cudaerr = denoise_knn_plane<Type, bit_depth>(&planeOutputY, &planeInputY, radius, strength, lerpC, weight_threshold, lerp_threshold, stream);
    if (cudaerr != hipSuccess) {
        return cudaerr;
    }
    cudaerr = denoise_knn_plane<Type, bit_depth>(&planeOutputU, &planeInputU, radius, strength, lerpC, weight_threshold, lerp_threshold, stream);
    if (cudaerr != hipSuccess) {
        return cudaerr;
    }
    cudaerr = denoise_knn_plane<Type, bit_depth>(&planeOutputV, &planeInputV, radius, strength, lerpC, weight_threshold, lerp_threshold, stream);
    if (cudaerr != hipSuccess) {
        return cudaerr;
    }
    return cudaerr;
}

NVEncFilterDenoiseKnn::NVEncFilterDenoiseKnn() : m_bInterlacedWarn(false) {
    m_name = _T("knn");
}

NVEncFilterDenoiseKnn::~NVEncFilterDenoiseKnn() {
    close();
}

RGY_ERR NVEncFilterDenoiseKnn::init(shared_ptr<NVEncFilterParam> pParam, shared_ptr<RGYLog> pPrintMes) {
    RGY_ERR sts = RGY_ERR_NONE;
    m_pLog = pPrintMes;
    auto pKnnParam = std::dynamic_pointer_cast<NVEncFilterParamDenoiseKnn>(pParam);
    if (!pKnnParam) {
        AddMessage(RGY_LOG_ERROR, _T("Invalid parameter type.\n"));
        return RGY_ERR_INVALID_PARAM;
    }
    //パラメータチェック
    if (pKnnParam->frameOut.height <= 0 || pKnnParam->frameOut.width <= 0) {
        AddMessage(RGY_LOG_ERROR, _T("Invalid parameter.\n"));
        return RGY_ERR_INVALID_PARAM;
    }
    if (pKnnParam->knn.radius <= 0) {
        AddMessage(RGY_LOG_ERROR, _T("radius must be a positive value.\n"));
        return RGY_ERR_INVALID_PARAM;
    }
    if (pKnnParam->knn.radius > KNN_RADIUS_MAX) {
        AddMessage(RGY_LOG_ERROR, _T("radius must be <= %d.\n"), KNN_RADIUS_MAX);
        return RGY_ERR_INVALID_PARAM;
    }
    if (pKnnParam->knn.strength < 0.0 || 1.0 < pKnnParam->knn.strength) {
        AddMessage(RGY_LOG_ERROR, _T("strength should be 0.0 - 1.0.\n"));
        return RGY_ERR_INVALID_PARAM;
    }
    if (pKnnParam->knn.lerpC < 0.0 || 1.0 < pKnnParam->knn.lerpC) {
        AddMessage(RGY_LOG_ERROR, _T("lerpC should be 0.0 - 1.0.\n"));
        return RGY_ERR_INVALID_PARAM;
    }
    if (pKnnParam->knn.lerp_threshold < 0.0 || 1.0 < pKnnParam->knn.lerp_threshold) {
        AddMessage(RGY_LOG_ERROR, _T("th_lerp should be 0.0 - 1.0.\n"));
        return RGY_ERR_INVALID_PARAM;
    }
    if (pKnnParam->knn.weight_threshold < 0.0 || 1.0 < pKnnParam->knn.weight_threshold) {
        AddMessage(RGY_LOG_ERROR, _T("th_weight should be 0.0 - 1.0.\n"));
        return RGY_ERR_INVALID_PARAM;
    }

    sts = AllocFrameBuf(pKnnParam->frameOut, 1);
    if (sts != RGY_ERR_NONE) {
        AddMessage(RGY_LOG_ERROR, _T("failed to allocate memory: %s.\n"), get_err_mes(sts));
        return sts;
    }
    for (int i = 0; i < RGY_CSP_PLANES[pParam->frameOut.csp]; i++) {
        pKnnParam->frameOut.pitch[i] = m_frameBuf[0]->frame.pitch[i];
    }

    setFilterInfo(pParam->print());
    m_param = pParam;
    return sts;
}

tstring NVEncFilterParamDenoiseKnn::print() const {
    return knn.print();
}

RGY_ERR NVEncFilterDenoiseKnn::run_filter(const RGYFrameInfo *pInputFrame, RGYFrameInfo **ppOutputFrames, int *pOutputFrameNum, hipStream_t stream) {
    RGY_ERR sts = RGY_ERR_NONE;

    if (pInputFrame->ptr[0] == nullptr) {
        return sts;
    }

    *pOutputFrameNum = 1;
    if (ppOutputFrames[0] == nullptr) {
        auto pOutFrame = m_frameBuf[m_nFrameIdx].get();
        ppOutputFrames[0] = &pOutFrame->frame;
        m_nFrameIdx = (m_nFrameIdx + 1) % m_frameBuf.size();
    }
    ppOutputFrames[0]->picstruct = pInputFrame->picstruct;
    if (interlaced(*pInputFrame)) {
        return filter_as_interlaced_pair(pInputFrame, ppOutputFrames[0], hipStreamDefault);
    }
    const auto memcpyKind = getCudaMemcpyKind(pInputFrame->mem_type, ppOutputFrames[0]->mem_type);
    if (memcpyKind != hipMemcpyDeviceToDevice) {
        AddMessage(RGY_LOG_ERROR, _T("only supported on device memory.\n"));
        return RGY_ERR_INVALID_PARAM;
    }
    if (m_param->frameOut.csp != m_param->frameIn.csp) {
        AddMessage(RGY_LOG_ERROR, _T("csp does not match.\n"));
        return RGY_ERR_INVALID_PARAM;
    }
    auto pKnnParam = std::dynamic_pointer_cast<NVEncFilterParamDenoiseKnn>(m_param);
    if (!pKnnParam) {
        AddMessage(RGY_LOG_ERROR, _T("Invalid parameter type.\n"));
        return RGY_ERR_INVALID_PARAM;
    }

    static const std::map<RGY_CSP, decltype(denoise_knn_frame<uint8_t, 8>)*> denoise_list = {
        { RGY_CSP_YV12,      denoise_knn_frame<uint8_t,   8> },
        { RGY_CSP_YV12_16,   denoise_knn_frame<uint16_t, 16> },
        { RGY_CSP_YUV444,    denoise_knn_frame<uint8_t,   8> },
        { RGY_CSP_YUV444_16, denoise_knn_frame<uint16_t, 16> },
    };
    if (denoise_list.count(pInputFrame->csp) == 0) {
        AddMessage(RGY_LOG_ERROR, _T("unsupported csp %s.\n"), RGY_CSP_NAMES[pInputFrame->csp]);
        return RGY_ERR_UNSUPPORTED;
    }
    sts = err_to_rgy(denoise_list.at(pInputFrame->csp)(ppOutputFrames[0], pInputFrame, pKnnParam->knn.radius, pKnnParam->knn.strength, pKnnParam->knn.lerpC, pKnnParam->knn.weight_threshold, pKnnParam->knn.lerp_threshold, stream));
    if (sts != RGY_ERR_NONE) {
        AddMessage(RGY_LOG_ERROR, _T("error at knn(%s): %s.\n"),
            RGY_CSP_NAMES[pInputFrame->csp],
            get_err_mes(sts));
        return sts;
    }
    return sts;
}

void NVEncFilterDenoiseKnn::close() {
    m_frameBuf.clear();
    m_bInterlacedWarn = false;
}
